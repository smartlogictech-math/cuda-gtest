/**
 * @file TEST_vadd.cu
 * @author zhe.zhang
 * @date 2025-03-18 16:20:18
 * @brief
 * @attention
 */

#include "vadd.h"
#include "testsuite_vadd.h"

#include <iostream>
#include "common.h"

TEST_F(VaddTestsuite, length_1024)
{
    const uint32_t N = 1024;
    float *h_A, *h_B, *h_C;

    CHECK(hipHostMalloc(&h_A, N * sizeof(float)));
    CHECK(hipHostMalloc(&h_B, N * sizeof(float)));
    CHECK(hipHostMalloc(&h_C, N * sizeof(float)));

    for (uint32_t i = 0; i < N; i++)
    {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void **)&d_A, N * sizeof(float)));
    CHECK(hipMalloc((void **)&d_B, N * sizeof(float)));
    CHECK(hipMalloc((void **)&d_C, N * sizeof(float)));

    CHECK(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t eStart, eEnd;
    hipEventCreate(&eStart);
    hipEventCreate(&eEnd);
    hipEventRecord(eStart);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(eEnd);
    hipEventSynchronize(eEnd);
    float time;
    hipEventElapsedTime(&time, eStart, eEnd);
    std::cout << "Elapesd time: " << time << "ms" << std::endl;

    CHECK(hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // 验证结果
    bool correct = true;
    for (uint32_t i = 0; i < N; i++)
    {
        if (h_C[i] != h_A[i] + h_B[i])
        {
            correct = false;
            std::cout << "Error at index " << i << ": " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            break;
        }
    }

    if (correct)
    {
        std::cout << "Vector addition successful!" << std::endl;
    }

    CHECK(hipEventDestroy(eStart));
    CHECK(hipEventDestroy(eEnd));
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(h_C));

    ASSERT_TRUE(correct);
}