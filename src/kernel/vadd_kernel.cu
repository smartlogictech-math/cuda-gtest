/**
 * @file vadd_kernel.cu
 * @author zhe.zhang
 * @date 2025-03-23 17:33:29
 * @brief 
 * @attention 
 */

#include <hip/hip_runtime.h>

__global__ void vadd_kernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void launch_vadd(const float* d_a, const float* d_b, float* d_c, int n, hipStream_t stream) {
    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);
    vadd_kernel<<<grid, block, 0, stream>>>(d_a, d_b, d_c, n);
}